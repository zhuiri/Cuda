
#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>
//#include <stdlib.h>
#include <string.h>
#include <time.h>

//__shared__ int ipiv[3];
__shared__ int indxc[3],indxr[3];


template<typename Typeval>
__device__ void Swap(Typeval &a,Typeval &b)
//void Swap(Typeval &a,Typeval &b)
{
	Typeval temp;
	temp=a;
	a=b;
	b=temp;
}

__global__ void kernel(double *a)
{
	/*b[0]=2*b[0];
	a[10]=a[10]+b[0];*/

	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;

	int offset=x+y*blockDim.x*gridDim.x;

	__shared__ float shared[32][32];

	const float period=128.0f;

	
	shared[threadIdx.x][threadIdx.y]=x+y;
	__syncthreads();

	
	a[offset]=255*shared[threadIdx.x][threadIdx.y];
	
	__syncthreads();
	

	


	int aa=0;



}

__global__ void fixRow(double*matrix,double*b,int size,int rowId)
{
	__shared__ double Ri[512];
	__shared__ double Bi[100];
	__shared__ double Aii;
	int colId=threadIdx.x;
	Ri[colId]=matrix[size*rowId+colId];//matrix[size*rowId+colId];

	Bi[colId]=b[size*rowId+0];

	Aii=matrix[size*rowId+rowId];//the diagonal element for ith row
	__syncthreads();

	Ri[colId]=Ri[colId]/Aii;
	matrix[size*rowId+colId]=Ri[colId];

	Bi[colId]=Bi[colId]/Aii;
	b[size*rowId+0]=Bi[colId];
}

__global__ void fixColumn(double *matrix,double *b,int size,int colId)
{
	int i=threadIdx.x;
	int j=blockIdx.x;

	__shared__ double col[512];
	
	__shared__ double AcolIdj;
	__shared__ double BcolIdj;
	__shared__ double colj[512];
	__shared__ double Bj[100];

	col[i]=matrix[i*size+colId];

	if(col[i]!=0)
	{
		colj[i]=matrix[i*size+j];
		Bj[i]=b[i*size+j];

		AcolIdj=matrix[colId*size+j];
		BcolIdj=b[colId*size+j];
		if(i!=colId)
		{
			colj[i]=colj[i]-AcolIdj*col[i];
			Bj[i]=Bj[i]-BcolIdj*col[i];
		}
		matrix[i*size+j]=colj[i];
		b[i*size+j]=Bj[i];

	}
}





extern "C" int
 runGauss(int MatrixSize,double *b,double**a )
{

	int vectorsize=30;
	//int MatrixSize=100;

	double *a_new,*b_new;
	
	a_new=new double[vectorsize];
	b_new=new double[vectorsize];
	//b=new float[vectorsize];
	//ipiv=new int[3];
	for(int i=0;i<MatrixSize;i++)
	{
		b_new[i*MatrixSize]=b[i];
		for(int j=0;j<MatrixSize;j++)
			a_new[i*MatrixSize+j]=a[i][j];
	}
	

	
	 
	 double *a_device,*b_device;
	
	 hipMalloc((void**)&a_device,vectorsize*sizeof(double));
	 hipMalloc((void**)&b_device,vectorsize*sizeof(double));
	 
	
	
	 hipEvent_t start,stop;
	 hipEventCreate(&start);
	 hipEventCreate(&stop);
	 hipEventRecord(start,0);
	 /*clock_t start,end;
	 start = clock();*/
	
		 hipMemcpy(a_device,a_new,vectorsize*sizeof(double),hipMemcpyHostToDevice);
	     hipMemcpy(b_device,b_new,vectorsize*sizeof(double),hipMemcpyHostToDevice);
		 for(int i=0;i<MatrixSize;i++)
		 {
			 fixRow<<<1,MatrixSize>>>(a_device,b_device,MatrixSize,i);	
			 fixColumn<<<MatrixSize,MatrixSize>>>(a_device,b_device,MatrixSize,i);
		 }
		 
	
	

	 hipEventRecord(stop,0);
	 hipEventSynchronize(stop);
	 float elapseTime;
	 hipEventElapsedTime(&elapseTime,start,stop);
	
	 hipMemcpy(a_new,a_device,vectorsize*sizeof(double),hipMemcpyDeviceToHost);
	 hipMemcpy(b_new,b_device,vectorsize*sizeof(double),hipMemcpyDeviceToHost);
	 for(int i=0;i<MatrixSize*MatrixSize;i++)
		 printf("%d-%5.3f\n",i+1,a_new[i]);


	 for(int i=0;i<MatrixSize;i++)
	 {
		 b[i]=b_new[i*MatrixSize];
		for(int j=0;j<MatrixSize;j++)
			a[i][j]=a_new[i*MatrixSize+j];
	 }

	 hipFree((void*)a_device);
	 hipFree((void*)b_device);

	 free(a_new);
	 free(b_new);
	 
	return 0;
}